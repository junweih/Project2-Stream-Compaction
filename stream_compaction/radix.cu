#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "radix.h"

namespace StreamCompaction {
    namespace RadixSort
    {
        using StreamCompaction::Common::PerformanceTimer;

        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernalCheckStop(int n, const int* idata, int* stop)
        {
            int index = threadIdx.x + (blockDim.x * blockIdx.x);
            if (index >= n - 1) return;

            if (idata[index] > idata[index + 1]) (*stop) = 1;
        }

        __global__ void kernalRadixMapToBoolean(int n, int k, int* label, const int* idata, int* skip) {
            int index = threadIdx.x + (blockDim.x * blockIdx.x);
            if (index >= n) return;

            int num = idata[index];
            int result = 1 - ((num & (1 << k)) >> k);
            if (k == 0 || result != ((num & (1 << (k - 1))) != 0 ? 0 : 1))
            {
                *skip = 1;
            }
            label[index] = result;
        }
        __global__ void kernalRadixScattering(int n, int k, int start, int* odata, const int* idata, const int* label)
        {
            int index = threadIdx.x + (blockDim.x * blockIdx.x);
            if (index >= n) return;

            bool result = ((idata[index] & (1 << k)) != 0 ? 1 : 0);
            if (result)
            {
                odata[start + index - label[index]] = idata[index];
            }
            else
            {
                odata[label[index]] = idata[index];
            }
        }

         void sort(int n, int* odata, int* idata) {
             int pot_length = n;// power-of-two length;

             int* dev_read;
             int* dev_write;
             int* dev_label;
             int* dev_number;

             hipMalloc((void**)&dev_read, pot_length * sizeof(int));
             checkCUDAError("hipMalloc dev_read failed!");
             hipMalloc((void**)&dev_write, pot_length * sizeof(int));
             checkCUDAError("hipMalloc dev_write failed!");
             hipMalloc((void**)&dev_label, pot_length * sizeof(int));
             checkCUDAError("hipMalloc dev_label failed!");
             hipMalloc((void**)&dev_number, sizeof(int));
             checkCUDAError("hipMalloc dev_number failed!");

             hipMemset(dev_read, (1 << 8) - 1, pot_length * sizeof(int));
             hipMemcpy(dev_read, idata, n * sizeof(int), hipMemcpyHostToDevice);
             checkCUDAError("Memcpy idata(host) to dev_read(device) failed!");
             timer().startGpuTimer();
             for (int i = 0; i < 32; ++i)
             {
                 // check whether to stop
                 hipMemset(dev_number, 0, sizeof(int));
                 kernalCheckStop << < (pot_length + blockSize - 1) / blockSize, blockSize >> > (pot_length, dev_read, dev_number);
                 int stop;
                 hipMemcpy(&stop, dev_number, sizeof(int), hipMemcpyDeviceToHost);
                 if (stop == 0) break;

                 // label and check whether to skip this bit
                 hipMemset(dev_number, 0, sizeof(int));
                 kernalRadixMapToBoolean << < (pot_length + blockSize - 1) / blockSize, blockSize >> > (pot_length, i, dev_label, dev_read, dev_number);
                 checkCUDAError("Luanch kernalRadixMapToBoolean failed!");

                 int skip;
                 hipMemcpy(&skip, dev_number, sizeof(int), hipMemcpyDeviceToHost);
                 if (skip == 0) continue;

                 // read the last number of label_1 back
                 int last_num;
                 hipMemcpy(&last_num, dev_label + pot_length - 1, sizeof(int), hipMemcpyDeviceToHost);

                 //Efficient::EfficientParallelScan(pot_length, dev_label);
                 thrust::device_ptr<int> thrust_dev_label(dev_label);
                 thrust::exclusive_scan(thrust_dev_label, thrust_dev_label + pot_length, dev_label);

                 int start_index;
                 hipMemcpy(&start_index, dev_label + pot_length - 1, sizeof(int), hipMemcpyDeviceToHost);
                 start_index += last_num;

                 kernalRadixScattering << < (pot_length + blockSize - 1) / blockSize, blockSize >> > (pot_length, i, start_index, dev_write, dev_read, dev_label);
                 checkCUDAError("Luanch kernalRadixMapToBoolean failed!");

                 std::swap(dev_write, dev_read);
             }
             timer().endGpuTimer();
             hipMemcpy(odata, dev_read, n * sizeof(int), hipMemcpyDeviceToHost);

             hipFree(dev_read);
             hipFree(dev_write);
             hipFree(dev_label);
             hipFree(dev_number);
        }
    }
}